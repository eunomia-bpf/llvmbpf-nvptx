#include <atomic>
#include <chrono>
#include <csignal>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <iostream>
#include <iterator>
#include <ostream>
#include <string>
#include <thread>
#include <vector>

/**
 * @brief Compile with clang++-17 -S ./test.cu -Wall --cuda-gpu-arch=sm_60 -O2
 * -L /usr/local/cuda/lib64/ -lcudart
 *
 */
enum class MapOperation { LOOKUP = 1, UPDATE = 2, DELETE = 3, NEXT_KEY = 4 };

union CallRequest {
	struct {
		char key[1 << 30];
	} map_lookup;
	struct {
		char key[1 << 30];
		char value[1 << 30];
		uint64_t flags;
	} map_update;
	struct {
		char key[1 << 30];
	} map_delete;
};

union CallResponse {
	struct {
		int result;
	} map_update, map_delete;
	struct {
		const void *value;
	} map_lookup;
};
/**
 * 我们在这块结构体里放两个标志位和一个简单的参数字段
 * - flag1: device -> host 的信号，“我有请求要处理”
 * - flag2: host   -> device 的信号，“我处理完了”
 * - paramA: 设备端写入的参数，让主机端使用
 */
struct SharedMem {
	int flag1;
	int flag2;
	int occupy_flag;
	int request_id;
	long map_id;
	CallRequest req;
	CallResponse resp;
	uint64_t time_sum[8];
};

struct MapBasicInfo {
	bool enabled;
	int key_size;
	int value_size;
	int max_entries;
};

__device__ __forceinline__ uint64_t read_globaltimer()
{
	uint64_t timestamp;
	asm volatile("mov.u64 %0, %%globaltimer;" : "=l"(timestamp));
	return timestamp;
}

__constant__ uintptr_t constData;
__constant__ MapBasicInfo map_info[256];
extern "C" __device__ void spin_lock(volatile int *lock)
{
	while (atomicCAS((int *)lock, 0, 1) == 1) {
		// 自旋等待锁变为可用
	}
	// printf("lock acquired by %d\n", threadIdx.x + blockIdx.x *
	// blockDim.x);
}

extern "C" __device__ void spin_unlock(int *lock)
{
	atomicExch(lock, 0); // 将锁标志重置为 0
	// printf("lock released by %d\n", threadIdx.x + blockIdx.x *
	// blockDim.x);
}
extern "C" __device__ CallResponse make_map_call(long map_id, int req_id)
{
	SharedMem *g_data = (SharedMem *)constData;
	// printf("make_map_call at %d, constdata=%lx\n",
	//        threadIdx.x + blockIdx.x * blockDim.x, (uintptr_t)g_data);
	auto start_time = read_globaltimer();
	spin_lock(&g_data->occupy_flag);
	// 准备要写入的参数值
	int val = 42; // 这里就写一个固定值，示例用
	// g_data->req = req;
	g_data->request_id = req_id;
	g_data->map_id = map_id;
	// printf("making call for %d\n", req_id);
	// 在内联PTX里演示 store/load + acquire/release + 自旋
	asm volatile(
		".reg .pred p0;                   \n\t" // 声明谓词寄存器
		"membar.sys;                      \n\t" // 内存屏障
							// 设置 flag1 = 1 (替代
							// st.global.rel.u32)
		"st.global.u32 [%1], 1;           \n\t"
		// 自旋等待 flag2 == 1 (替代 ld.global.acq.u32)
		"spin_wait:                       \n\t"
		"membar.sys;                      \n\t"
		"ld.global.u32 %0, [%2];          \n\t" // 读取 flag2
		"setp.eq.u32 p0, %0, 0;           \n\t" // 比较值
		"@p0 bra spin_wait;               \n\t" // 谓词分支
							// 若跳出循环，复位
							// flag2 = 0
		"st.global.u32 [%2], 0;           \n\t"
		"membar.sys;                      \n\t"
		:
		: "r"(val), "l"(&g_data->flag1), "l"(&g_data->flag2)
		: "memory");
	CallResponse resp = g_data->resp;

	spin_unlock(&g_data->occupy_flag);
	auto end_time = read_globaltimer();
	if (req_id < 8) {
		atomicAdd((unsigned long long *)&g_data->time_sum[req_id],
			  end_time - start_time);
	}
	return resp;
}

extern "C" __device__ inline void simple_memcpy(void *dst, void *src, int sz)
{
	for (int i = 0; i < sz; i++)
		((char *)dst)[i] = ((char *)src)[i];
}

extern "C" __noinline__ __device__ uint64_t _bpf_helper_ext_0001(
	uint64_t map, uint64_t key, uint64_t a, uint64_t b, uint64_t c)
{
	SharedMem *global_data = (SharedMem *)constData;
	auto &req = global_data->req;
	// CallRequest req;
	const auto &map_info = ::map_info[map >> 32];
	// printf("helper1 map %ld keysize=%d valuesize=%d\n", map,
	//        map_info.key_size, map_info.value_size);
	simple_memcpy(&req.map_lookup.key, (void *)(uintptr_t)key,
		      map_info.key_size);

	CallResponse resp = make_map_call((long)map, (int)MapOperation::LOOKUP);

	return (uintptr_t)resp.map_lookup.value;
}

extern "C" __noinline__ __device__ uint64_t _bpf_helper_ext_0002(
	uint64_t map, uint64_t key, uint64_t value, uint64_t flags, uint64_t a)
{
	SharedMem *global_data = (SharedMem *)constData;
	auto &req = global_data->req;
	const auto &map_info = ::map_info[map >> 32];
	// printf("helper2 map %ld keysize=%d
	// valuesize=%d\n",map,map_info.key_size,map_info.value_size);
	simple_memcpy(&req.map_update.key, (void *)(uintptr_t)key,
		      map_info.key_size);
	simple_memcpy(&req.map_update.value, (void *)(uintptr_t)value,
		      map_info.value_size);
	req.map_update.flags = (uintptr_t)flags;

	CallResponse resp = make_map_call((long)map, (int)MapOperation::UPDATE);
	return resp.map_update.result;
}

extern "C" __noinline__ __device__ uint64_t _bpf_helper_ext_0003(
	uint64_t map, uint64_t key, uint64_t a, uint64_t b, uint64_t c)
{
	SharedMem *global_data = (SharedMem *)constData;
	auto &req = global_data->req;
	const auto &map_info = ::map_info[map >> 32];
	// printf("helper3 map %ld keysize=%d
	// valuesize=%d\n",map,map_info.key_size,map_info.value_size);
	simple_memcpy(&req.map_delete.key, (void *)(uintptr_t)key,
		      map_info.key_size);
	CallResponse resp = make_map_call((long)map, (int)MapOperation::DELETE);
	return resp.map_delete.result;
}

extern "C" __global__ void bpf_main(void *mem, size_t sz)
{
	printf("kernel function entered, mem=%lx, memsz=%ld\n", (uintptr_t)mem,
	       sz);
	char buf[16] = "aaa";
	printf("setup function, const data=%lx\n", constData);
	auto result = _bpf_helper_ext_0001(1ull << 32, (uintptr_t)buf, 0, 0, 0);
	_bpf_helper_ext_0002(1ull << 32, (uintptr_t)buf, (uintptr_t)buf, 0, 0);
	_bpf_helper_ext_0003(1ull << 32, (uintptr_t)buf, 0, 0, 0);
	printf("call done\n");
	printf("got response %d at %d\n", *(int *)result,
	       threadIdx.x + blockIdx.x * blockDim.x);
	*(int *)mem = 123;
}

static std::atomic<bool> should_exit;
void signal_handler(int)
{
	should_exit.store(true);
}
int main()
{
	signal(SIGINT, signal_handler);

	// 1. 先在主机上分配一块普通内存
	SharedMem *hostMem = (SharedMem *)malloc(sizeof(SharedMem));
	if (!hostMem) {
		std::cerr << "Failed to allocate hostMem\n";
		return -1;
	}

	// 2. 注册成 pinned memory (可被GPU直接访问)
	hipError_t err = hipHostRegister(hostMem, sizeof(SharedMem),
					   hipHostRegisterMapped);
	if (err != hipSuccess) {
		std::cerr
			<< "hipHostRegister error: " << hipGetErrorString(err)
			<< "\n";
		free(hostMem);
		return -1;
	}

	// 3. 获取对应的设备指针(这样DeviceKernel就能直接访问这个地址)
	SharedMem *devPtr = nullptr;
	err = hipHostGetDevicePointer((void **)&devPtr, (void *)hostMem, 0);
	if (err != hipSuccess) {
		std::cerr << "hipHostGetDevicePointer error: "
			  << hipGetErrorString(err) << "\n";
		hipHostUnregister(hostMem);
		free(hostMem);
		return -1;
	}
	printf("dev ptr should be %lx, host ptr is %lx\n", (uintptr_t)devPtr,
	       (uintptr_t)hostMem);
	err = hipMemcpyToSymbol(HIP_SYMBOL(constData), &devPtr, sizeof(SharedMem *));
	if (err != hipSuccess) {
		std::cerr << "hipMemcpyToSymbol error: "
			  << hipGetErrorString(err) << "\n";
		hipHostUnregister(hostMem);
		free(hostMem);
		return -1;
	}
	int buf = 11223344;
	err = hipHostRegister((void *)&buf, sizeof(buf),
			       hipHostRegisterMapped);
	if (err != hipSuccess) {
		std::cerr << "cudaHostRegister(2) error: "
			  << hipGetErrorString(err) << " " << err << "\n";
		hipHostUnregister(hostMem);
		free(hostMem);
		return -1;
	}
	char *devPtrStr = nullptr;
	err = hipHostGetDevicePointer((void **)&devPtrStr, (void *)&buf, 0);
	if (err != hipSuccess) {
		std::cerr << "cudaHostGetDevicePointer(2) error: "
			  << hipGetErrorString(err) << "\n";
		hipHostUnregister(hostMem);
		free(hostMem);
		return -1;
	}
	// 初始化标志位
	memset(hostMem, 0, sizeof(*hostMem));
	// 4. 启动一个线程, 模拟host侧的处理逻辑
	std::thread hostThread([&]() {
		std::cout << "[Host Thread] Start waiting...\n";

		// 这里简单用轮询，检测到flag1=1就处理
		while (!should_exit.load()) {
			if (hostMem->flag1 == 1) {
				// 清掉flag1防止重复处理
				hostMem->flag1 = 0;
				// 假设处理数据 paramA
				std::cout
					<< "[Host Thread] Got request: req_id="
					<< hostMem->request_id
					<< ", handling...\n";
				if (hostMem->request_id == 1) {
					std::cout << "call map_lookup="
						  << hostMem->req.map_lookup.key
						  << std::endl;
					// strcpy(hostMem->resp.map_lookup.value,
					//        "your value");
					hostMem->resp.map_lookup.value =
						devPtrStr;
				}
				// std::atomic_thread_fence(std::memory_order_seq_cst);

				// 处理完后, 把 flag2=1, 让设备端退出自旋
				hostMem->flag2 = 1;

				// 在实际开发中，可以加个内存栅栏，比如：
				std::atomic_thread_fence(
					std::memory_order_seq_cst);

				// 处理一次就退出本线程循环
				// break;
				std::cout << "handle done, timesum = "
					  << hostMem->time_sum[1] << std::endl;
			}

			// 为了演示，这里短暂休眠，避免100%占用CPU
			std::this_thread::sleep_for(
				std::chrono::milliseconds(10));
		}

		std::cout << "[Host Thread] Done.\n";
	});
	std::vector<MapBasicInfo> local_map_info(256);

	local_map_info[1].enabled = true;
	local_map_info[1].key_size = 16;
	local_map_info[1].value_size = 16;
	hipMemcpyToSymbol(HIP_SYMBOL(map_info), local_map_info.data(),
			   sizeof(MapBasicInfo) * local_map_info.size());
	// 5. 启动核函数 (只发1个block,1个thread做演示)
	bpf_main<<<1, 1>>>(hostMem, sizeof(*hostMem));

	// 等待核函数执行完毕
	hipDeviceSynchronize();

	// 等待host线程结束
	hostThread.join();

	// 6. 收尾：解绑 pinned memory 并释放
	hipHostUnregister(hostMem);
	free(hostMem);

	std::cout << "All done.\n";
	return 0;
}
